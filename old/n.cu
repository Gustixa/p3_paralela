#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <vector>
#include "image.h" // Cambiamos pgm.h a image.h
#include "utils.h"
#include "cpu_hough.h"
#include "stb_image.h"
#include <opencv2/opencv.hpp> // tener cuidado, falla a veces

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

// Declaración de memoria constante
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

// Kernel con sólo memoria global
__global__ void GPU_HoughTran_Global(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale) {
    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID >= w * h) return;

    int xCent = w / 2;
    int yCent = h / 2;

    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    if (pic[gloID] > 0) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            atomicAdd(&acc[rIdx * degreeBins + tIdx], 1);
        }
    }
}

// Kernel con memoria constante y global
__global__ void GPU_HoughTran_ConstGlobal(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale) {
    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID >= w * h) return;

    int xCent = w / 2;
    int yCent = h / 2;

    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    if (pic[gloID] > 0) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            atomicAdd(&acc[rIdx * degreeBins + tIdx], 1);
        }
    }
}

// Kernel con memoria constante, global y compartida
__global__ void GPU_HoughTran_ConstGlobalShared(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale) {
    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID >= w * h) return;

    int xCent = w / 2;
    int yCent = h / 2;

    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    __shared__ int localAcc[degreeBins * rBins];  // Memoria compartida para el acumulador
    if (threadIdx.x < degreeBins * rBins)
        localAcc[threadIdx.x] = 0;

    __syncthreads();

    if (pic[gloID] > 0) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            atomicAdd(&localAcc[rIdx * degreeBins + tIdx], 1);
        }
    }

    __syncthreads();
    if (threadIdx.x < degreeBins * rBins) {
        atomicAdd(&acc[threadIdx.x], localAcc[threadIdx.x]);
    }
}

// Función para ejecutar el experimento con diferentes configuraciones
void runExperiment(int kernelType, unsigned char *d_in, int *d_hough, int w, int h, float rMax, float rScale, int numTrials) {
    std::vector<float> times;
    for (int i = 0; i < numTrials; i++) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start); // Inicia el temporizador
        
        // Selecciona y ejecuta el kernel basado en kernelType
        if (kernelType == 0) {
            GPU_HoughTran_Global<<<ceil(w * h / 256.0), 256>>>(d_in, w, h, d_hough, rMax, rScale);
        } else if (kernelType == 1) {
            GPU_HoughTran_ConstGlobal<<<ceil(w * h / 256.0), 256>>>(d_in, w, h, d_hough, rMax, rScale);
        } else if (kernelType == 2) {
            GPU_HoughTran_ConstGlobalShared<<<ceil(w * h / 256.0), 256>>>(d_in, w, h, d_hough, rMax, rScale);
        }
        
        hipEventRecord(stop); // Detiene el temporizador
        hipEventSynchronize(stop); // Sincroniza para asegurarse de que el kernel ha terminado

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); // Calcula el tiempo de ejecución
        times.push_back(milliseconds); // Guarda el tiempo en la lista

        hipEventDestroy(start); // Limpia los eventos de CUDA
        hipEventDestroy(stop);
    }
    
    // Calcula el tiempo promedio
    float total = 0;
    for (float time : times) {
        total += time;
    }
    float avgTime = total / numTrials;
    
    // Imprime los tiempos y el promedio
    std::cout << "Kernel Type " << kernelType << ": Average Time = " << avgTime << " ms\n";
    for (int i = 0; i < times.size(); i++) {
        std::cout << "  Trial " << i + 1 << ": " << times[i] << " ms\n";
    }
}

// Función para mostrar el acumulador de Hough
void displayHoughAccumulator(int *h_acc, int degreeBins, int rBins) {
    // Convertir el acumulador en una imagen
    cv::Mat houghImage(rBins, degreeBins, CV_32S, h_acc);

    // Imprimir los valores mínimos y máximos del acumulador
    double minVal, maxVal;
    cv::minMaxLoc(houghImage, &minVal, &maxVal);
    std::cout << "Min value: " << minVal << ", Max value: " << maxVal << std::endl;

    // Normalizar para que los valores vayan de 0 a 255
    cv::Mat houghImageNorm;
    cv::normalize(houghImage, houghImageNorm, 0, 255, cv::NORM_MINMAX, CV_8U);

    // Crear una ventana y redimensionarla
    std::string windowName = "Hough Transform Accumulator";
    cv::namedWindow(windowName, cv::WINDOW_NORMAL);  // Usa cv::WINDOW_NORMAL para permitir el redimensionamiento
    cv::resizeWindow(windowName, 800, 600);          // Ajusta el tamaño de la ventana a 800x600 (o el tamaño que prefieras)

    // Mostrar la imagen del acumulador de Hough
    cv::imshow(windowName, houghImageNorm);
    cv::waitKey(0);
}

// Función principal
int main(int argc, char **argv) {
    // Inicialización de la imagen
    Image inImg(argv[1]);
    if (!inImg.pixels) {
        std::cerr << "Error: No se pudo cargar la imagen.\n";
        return -1;
    }

    int w = inImg.x_dim;
    int h = inImg.y_dim;

    // Pre-compute and copy cos/sin values
    float *pcCos, *pcSin;
    precomputeTrig(degreeBins, radInc, &pcCos, &pcSin);

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins);

    // Configuración de memoria en la GPU
    unsigned char *d_in;
    int *d_hough;
    hipMalloc((void **) &d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **) &d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, inImg.pixels, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

    int numTrials = 10;
    
    std::cout << "Running experiments...\n";
    
    // Ejecutar experimentos para cada configuración de memoria
    runExperiment(0, d_in, d_hough, w, h, rMax, rScale, numTrials); // Solo Global
    runExperiment(1, d_in, d_hough, w, h, rMax, rScale, numTrials); // Constante y Global
    runExperiment(2, d_in, d_hough, w, h, rMax, rScale, numTrials); // Constante, Global y Compartida

    // Copiar el acumulador desde la memoria del dispositivo a la memoria del host
    // int *h_hough = (int *)malloc(sizeof(int) * degreeBins * rBins);
    // hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    // // Desplegar el acumulador de Hough
    // displayHoughAccumulator(h_hough, degreeBins, rBins);

    // Limpieza
    cleanup(d_in, d_hough, pcCos, pcSin, nullptr, nullptr);
    printf("Done!\n");

    return 0;
}
